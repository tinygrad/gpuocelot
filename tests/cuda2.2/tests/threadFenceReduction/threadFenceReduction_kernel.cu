/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/*
    Parallel reduction kernels
*/

#ifndef _REDUCE_KERNEL_H_
#define _REDUCE_KERNEL_H_

#ifdef __DEVICE_EMULATION__
#define EMUSYNC __syncthreads()
#else
#define EMUSYNC
#endif

#include <hip/device_functions.h>

/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n/2 threads
    - only works for power-of-2 arrays

    This version adds multiple elements per thread sequentially.  This reduces the overall
    cost of the algorithm while keeping the work complexity O(n) and the step complexity O(log n).
    (Brent's Theorem optimization)

    See the CUDA SDK "reduction" sample for more information.
*/

template <unsigned int blockSize>
__device__ void
reduceBlock(float *sdata, const unsigned int tid)
{
    // do reduction in shared mem
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid <  64) { sdata[tid] += sdata[tid +  64]; } __syncthreads(); }

#ifndef __DEVICE_EMULATION__
    if (tid < 32)
#endif
    {
        if (blockSize >=  64) { sdata[tid] += sdata[tid + 32]; EMUSYNC; }
        if (blockSize >=  32) { sdata[tid] += sdata[tid + 16]; EMUSYNC; }
        if (blockSize >=  16) { sdata[tid] += sdata[tid +  8]; EMUSYNC; }
        if (blockSize >=   8) { sdata[tid] += sdata[tid +  4]; EMUSYNC; }
        if (blockSize >=   4) { sdata[tid] += sdata[tid +  2]; EMUSYNC; }
        if (blockSize >=   2) { sdata[tid] += sdata[tid +  1]; EMUSYNC; }
    }
}

template <unsigned int blockSize, bool nIsPow2>
__device__ void
reduceBlocks(const float *g_idata, float *g_odata, unsigned int n)
{
    extern __shared__ float sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;
    sdata[tid] = 0;

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {         
        sdata[tid] += g_idata[i];
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n) 
            sdata[tid] += g_idata[i+blockSize];  
        i += gridSize;
    } 

    __syncthreads();

    // do reduction in shared mem
    reduceBlock<blockSize>(sdata, tid);
   
    // write result for this block to global mem 
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


template <unsigned int blockSize, bool nIsPow2>
__global__ void
reduceMultiPass(const float *g_idata, float *g_odata, unsigned int n)
{
    reduceBlocks<blockSize, nIsPow2>(g_idata, g_odata, n);
}

// Global variable used by reduceSinglePass to count how many blocks have finished
__device__ unsigned int retirementCount = 0;


// This reduction kernel reduces an arbitrary size array in a single kernel invocation
// It does so by keeping track of how many blocks have finished.  After each thread
// block completes the reduction of its own block of data, it "takes a ticket" by
// atomically incrementing a global counter.  If the ticket value is equal to the number
// of thread blocks, then the block holding the ticket knows that it is the last block
// to finish.  This last block is responsible for summing the results of all the other
// blocks.
//
// In order for this to work, we must be sure that before a block takes a ticket, all 
// of its memory transactions have completed.  This is what __threadfence() does -- it
// blocks until the results of all outstanding memory transactions within the 
// calling thread are visible to all other threads.
//
// For more details on the reduction algorithm (notably the multi-pass approach), see
// the "reduction" sample in the CUDA SDK.
template <unsigned int blockSize, bool nIsPow2>
__global__ void reduceSinglePass(const float *g_idata, float *g_odata, unsigned int n)
{

    //
    // PHASE 1: Process all inputs assigned to this block
    //

    reduceBlocks<blockSize, nIsPow2>(g_idata, g_odata, n);

    //
    // PHASE 2: Last block finished will process all partial sums
    //

    if (gridDim.x > 1)
    {
        const unsigned int tid = threadIdx.x;
        __shared__ bool amLast;
        extern float __shared__ smem[];

        // wait until all outstanding memory instructions in this thread are finished
        __threadfence();

        // Thread 0 takes a ticket
        if( tid==0 )
        {
            unsigned int ticket = atomicInc(&retirementCount, gridDim.x);
            // If the ticket ID is equal to the number of blocks, we are the last block!
            amLast = (ticket == gridDim.x-1);
        }
        __syncthreads();

        // The last block sums the results of all other blocks
        if( amLast )
        {
            // load block results back into shared memory
            smem[tid] = (tid < gridDim.x) ? g_odata[tid] : 0;
            
            __syncthreads();
            
            reduceBlock<blockSize>(smem, tid);
            
            if( tid==0 )  
            {
                g_odata[0] = smem[0];
                
                // reset retirement count so that next run succeeds
                retirementCount = 0; 
            }
        }
    }
}

bool isPow2(unsigned int x)
{
    return ((x&(x-1))==0);
}


////////////////////////////////////////////////////////////////////////////////
// Wrapper function for kernel launch
////////////////////////////////////////////////////////////////////////////////
extern "C"
void reduce(int size, int threads, int blocks, float *d_idata, float *d_odata)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);
    int smemSize = threads * sizeof(float);

    // choose which of the optimized versions of reduction to launch
    if (isPow2(size))
    {
        switch (threads)
        {
        case 512:
            reduceMultiPass<512, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 256:
            reduceMultiPass<256, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 128:
            reduceMultiPass<128, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 64:
            reduceMultiPass< 64, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 32:
            reduceMultiPass< 32, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 16:
            reduceMultiPass< 16, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  8:
            reduceMultiPass<  8, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  4:
            reduceMultiPass<  4, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  2:
            reduceMultiPass<  2, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  1:
            reduceMultiPass<  1, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        }
    }
    else
    {
        switch (threads)
        {
        case 512:
            reduceMultiPass<512, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 256:
            reduceMultiPass<256, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 128:
            reduceMultiPass<128, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 64:
            reduceMultiPass< 64, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 32:
            reduceMultiPass< 32, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 16:
            reduceMultiPass< 16, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  8:
            reduceMultiPass<  8, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  4:
            reduceMultiPass<  4, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  2:
            reduceMultiPass<  2, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  1:
            reduceMultiPass<  1, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        }
    }
}

extern "C"
void reduceSinglePass(int size, int threads, int blocks, float *d_idata, float *d_odata)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);
    int smemSize = threads * sizeof(float);

    // choose which of the optimized versions of reduction to launch
    if (isPow2(size))
    {
        switch (threads)
        {
        case 512:
            reduceSinglePass<512, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 256:
            reduceSinglePass<256, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 128:
            reduceSinglePass<128, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 64:
            reduceSinglePass< 64, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 32:
            reduceSinglePass< 32, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case 16:
            reduceSinglePass< 16, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  8:
            reduceSinglePass<  8, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  4:
            reduceSinglePass<  4, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  2:
            reduceSinglePass<  2, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        case  1:
            reduceSinglePass<  1, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        }
    }
    else
    {
        switch (threads)
        {
            case 512:
                reduceSinglePass<512, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
            case 256:
                reduceSinglePass<256, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
            case 128:
                reduceSinglePass<128, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
            case 64:
                reduceSinglePass< 64, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
            case 32:
                reduceSinglePass< 32, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
            case 16:
                reduceSinglePass< 16, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
            case  8:
                reduceSinglePass<  8, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
            case  4:
                reduceSinglePass<  4, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
            case  2:
                reduceSinglePass<  2, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
            case  1:
                reduceSinglePass<  1, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
        }
}
}

#endif // #ifndef _REDUCE_KERNEL_H_